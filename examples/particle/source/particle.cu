#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

extern "C" {
#include "ido.h"
}



__global__  void  gpu_ppush1
// ====================================================================
//
// program    :  CUDA device code for the particle push calculation
//               by 4-stage Runge-Kutta time integration
//
// date       :  Jul 3, 2014
// programmer :  Michel Müller, Original by Dr. Takayuki Aoki
// place      :  Tokyo Institute of Technology
//
(
   FLOAT    *x,         /* x-coordinate of the particles             */
   FLOAT    *y,         /* y-coordinate of the particles             */
   FLOAT    *xn,        /* updated x-coordinate of the particles     */
   FLOAT    *yn,        /* updated y-coordinate of the particles     */
   FLOAT	time,
   FLOAT    time_factor,       /* time                                      */
   FLOAT    dt          /* time step interval                        */
)
// --------------------------------------------------------------------
{
	int   j,   jx,  jy;
	FLOAT  xg,  yg,  xtdt,  ytdt;

	jx = blockDim.x*blockIdx.x + threadIdx.x;
	jy = blockDim.y*blockIdx.y;
	j = gridDim.x*blockDim.x*jy + jx;

	xg = x[j];    yg = y[j];

	/* -------- original algorithm ----------------- */
	// xtdt = US(xg, yg, time)*dt;
	// ytdt = VS(xg, yg, time)*dt;
	/* -------- end of original algorithm ---------- */

	/* -------- optimized algorithm like on CPU ---------------- */
	FLOAT mpi_xg = M_PI * xg;
	FLOAT mpi_yg = M_PI * yg;
	FLOAT sin_xg = SIN_D(mpi_xg);
	FLOAT cos_xg = COS_D(mpi_xg);
	FLOAT sin_yg = SIN_D(mpi_yg);
	FLOAT cos_yg = COS_D(mpi_yg);

	xtdt = (-1) * time_factor * sin_xg * sin_xg * cos_yg  * sin_yg;
	ytdt = time_factor * cos_xg * sin_xg * sin_yg * sin_yg;
	/* -------- end of optimized algorithm --------- */

	xn[j] = xg + xtdt;
	yn[j] = yg + ytdt;
}

void  ppush
// ====================================================================
//
// purpos     :  Particle push by 4-stage Runge-Kutta time integration
//
// date       :  Jul 3, 2014
// programmer :  Michel Müller
// place      :  Tokyo Institute of Technology
//
(
   int      np,         /* number of the particles                   */
   FLOAT    *x,         /* x-coordinate of the particles             */
   FLOAT    *y,         /* y-coordinate of the particles             */
   FLOAT    *xn,        /* updated x-coordinate of the particles     */
   FLOAT    *yn,        /* updated y-coordinate of the particles     */
   FLOAT    time,       /* time                                      */
   FLOAT    dt          /* time step interval                        */
)
// --------------------------------------------------------------------
{


	FLOAT time_factor = 2.0f * cos(M_PI * time / TAU) * dt;
	dim3  Dg(np/128,1,1),  Db(128,1,1);
    gpu_ppush1<<< Dg, Db >>>(x,y,xn,yn,time, time_factor,dt);
    hipDeviceSynchronize();

}

void mainloop
// ====================================================================
//
// purpos     :  Particle push by 1-stage Runge-Kutta: time integration
//
// date       :  2012-5-8
// programmer :  Michel Müller, Original by Dr. Takayuki Aoki
// place      :  Tokyo Institute of Technology
//
(
   int      np,         /* number of the particles                   */
   FLOAT    *x,         /* x-coordinate of the particles             */
   FLOAT    *y,         /* y-coordinate of the particles             */
   FLOAT    *xn,        /* updated x-coordinate of the particles     */
   FLOAT    *yn,        /* updated y-coordinate of the particles     */
   FLOAT    time,       /* time                                      */
   FLOAT    dt          /* time step interval                        */
)
// --------------------------------------------------------------------
{
	int  icnt = 1;
	long long int numOfPointUpdates = 0;

	double start_time_total, start_computation_time, elapsed_time_total, elapsed_computation_time;
    start_time_total = omp_get_wtime();
    // unsigned int timer;

	FLOAT *x_d, *y_d, *xn_d, *yn_d;
	// FLOAT gpu_time;

	hipMalloc( (void**) &x_d,  np*sizeof(FLOAT));
	hipMalloc( (void**) &y_d,  np*sizeof(FLOAT));
	hipMalloc( (void**) &xn_d, np*sizeof(FLOAT));
	hipMalloc( (void**) &yn_d, np*sizeof(FLOAT));
	hipMemcpy(x_d,x,np*sizeof(FLOAT), hipMemcpyHostToDevice);
    hipMemcpy(y_d,y,np*sizeof(FLOAT), hipMemcpyHostToDevice);
	{
		start_computation_time = omp_get_wtime();
		// cutCreateTimer(&timer);
		// cutResetTimer(timer);
		// cutStartTimer(timer);

		do {  if(icnt % 500 == 0) aprint("time(%4d)=%7.5f\n",icnt,time + dt);

			  ppush(np,x_d,y_d,xn_d,yn_d,time,dt);
			  swap(&x_d,&xn_d);  swap(&y_d, &yn_d);
			  time += dt;
			  numOfPointUpdates += np;
		} while(icnt++ < 999999 && time < 8.0 - 0.5*dt);

		// cutStopTimer(timer);
     	// gpu_time = cutGetTimerValue(timer)*1.0e-03;
		elapsed_computation_time = omp_get_wtime() - start_computation_time;
   	}
   	hipMemcpy(x, x_d,np*sizeof(FLOAT), hipMemcpyDeviceToHost);
    hipMemcpy(y, y_d,np*sizeof(FLOAT), hipMemcpyDeviceToHost);
	elapsed_time_total = omp_get_wtime() - start_time_total;

	hipFree(x_d);
	hipFree(y_d);
	hipFree(xn_d);
	hipFree(yn_d);

	aprint("Elapsed Total Time (OMP timer)= %9.3e [sec]\n",elapsed_time_total);
	aprint("Elapsed Computation Time (OMP timer)= %9.3e [sec]\n",elapsed_computation_time);
	aprint("Performance= %7.2f [million point updates/sec]\n",((double)numOfPointUpdates)/elapsed_time_total*1.0e-06);
	// aprint("Elapsed Computation Time (CUDA timer)= %9.3e [sec]\n",gpu_time);
	printf("%9.3e,%7.2f,%9.3e\n", elapsed_computation_time, (double)numOfPointUpdates/elapsed_time_total*1.0e-06, elapsed_time_total);
}
